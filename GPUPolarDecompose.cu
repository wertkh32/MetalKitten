#pragma once
#include <hip/hip_runtime.h>

#define TOLERANCE 0.01

__device__ 
float oneNorm(const float A[3][3])
{
  float norm = 0.0;
  #pragma unroll 3
  for (int i=0; i<3; i++) 
  {
    float columnAbsSum = fabsf(A[0][i]) + fabsf(A[1][i]) + fabsf(A[2][i]);
    if (columnAbsSum > norm) 
      norm = columnAbsSum;
  }
  return norm;
}

__device__
float infNorm(const float A[3][3])
{
  float norm = 0.0;
  #pragma unroll 3
  for (int i=0; i<3; i++) 
  {
    float rowSum = fabsf(A[i][0]) + fabsf(A[i][1]) + fabsf(A[i][2]);
    if (rowSum > norm) 
      norm = rowSum;
  }
  return norm;
}

// cross product: c = a x b
__device__
void crossProduct(const float* a, const float* b, float* c)
{
  c[0] = a[1] * b[2] - a[2] * b[1];
  c[1] = a[2] * b[0] - a[0] * b[2];
  c[2] = a[0] * b[1] - a[1] * b[0];
}

// Input: M (3x3 mtx)
// Output: Q (3x3 rotation mtx), S (3x3 symmetric mtx)
__device__
void gpuComputePolarDecomposition(float Mk[3][3])
{
  //float Mk[3][3];
  float Ek[3][3];
  float det, M_oneNorm, M_infNorm, E_oneNorm;

  // Mk = M^T
  //for(int i=0; i<3; i++)
  //  for(int j=0; j<3; j++)
  //    Mk[i][j] = M[j][i];

  M_oneNorm = oneNorm(Mk); 
  M_infNorm = infNorm(Mk);

  do 
  {
    float MadjTk[3][3];
 
    // row 2 x row 3
    crossProduct(Mk[1], Mk[2], MadjTk[0]); 
    // row 3 x row 1
    crossProduct(Mk[2], Mk[0], MadjTk[1]);
    // row 1 x row 2
    crossProduct(Mk[0], Mk[1], MadjTk[2]);

    det = Mk[0][0] * MadjTk[0][0] + Mk[0][1] * MadjTk[0][1] + Mk[0][2] * MadjTk[0][2];
    if (det == 0.0) 
    {
      //printf("Warning (polarDecomposition) : zero determinant encountered.\n");
      break;
    }

    float MadjT_one = oneNorm(MadjTk); 
    float MadjT_inf = infNorm(MadjTk);

    float gamma = sqrtf(sqrtf((MadjT_one * MadjT_inf) / (M_oneNorm * M_infNorm)) / fabsf(det));
    float g1 = gamma * 0.5;
    float g2 = 0.5 / (gamma * det);

	#pragma unroll 3
    for(int i=0; i<3; i++)
		#pragma unroll 3
		for(int j=0; j<3; j++)
		{
			Ek[i][j] = Mk[i][j];
			Mk[i][j] = g1 * Mk[i][j] + g2 * MadjTk[i][j];
			Ek[i][j] -= Mk[i][j];
		}

    E_oneNorm = oneNorm(Ek);
    M_oneNorm = oneNorm(Mk);  
    M_infNorm = infNorm(Mk);
  }
  while ( E_oneNorm > M_oneNorm * TOLERANCE );

  // Q = Mk^T 
  //for(int i=0; i<3; i++)
  //  for(int j=0; j<3; j++)
  //    Q[i][j] = Mk[j][i];

  //return (det);
}
