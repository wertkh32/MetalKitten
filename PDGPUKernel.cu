#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"
#include "PDGPUKernel.cuh"
#include "GPUPolarDecompose.cu"

#define DAMPING 0.977
#define PD_ITERATIONS 5
#define CG_ITERATIONS 5

float* debugbuf;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		system("pause");
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

TetData* gpuptr_TetData;
NodeData*   gpuptr_NodeData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_fext;//dynamic
float*	 gpuptr_b;//dynamic
float*	 gpuptr_sn;
float*	 gpuptr_x;
float*	 gpuptr_x0;
float*	 gpuptr_mass;
char*	 gpuptr_allowed;

//for CG
float* gpuptr_CG_R;
float* gpuptr_CG_D;
float* gpuptr_CG_Q;
CGVars* gpuptr_CG_Vars;

//for debug
float* gpuptr_debug;


__host__
void checkCudaErrors(const char* comment)
{
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("\n%s\nCUDA error: %s\n", comment, hipGetErrorString(error));
		//exit(-1);
		system("pause");
	}
}

__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / TET_BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	HANDLE_ERROR( hipMalloc(&gpuptr_TetData, numblocksperele * sizeof(TetData)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_NodeData, numblockpernode * sizeof(NodeData)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_fext, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_sn, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_x, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_x0, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_mass, numnodes * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_allowed, numnodes * sizeof(char)) );

	HANDLE_ERROR( hipMalloc(&gpuptr_CG_R, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_CG_D, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_CG_Q, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_CG_Vars, sizeof(CGVars)) );

	debugbuf = (float*)malloc(numnodes * 3 * sizeof(float));
	HANDLE_ERROR( hipMalloc(&gpuptr_debug, numnodes * 3 * sizeof(float)) );

	checkCudaErrors("Allocation");
}

__host__
void
gpuUploadVars(TetData* gpuElements, NodeData* gpuNodes,float* x, float* x0, 
			  float* vt, float* extforces, float* mass, char* allowed, int numnodes, int numelements)
{
	int numblocksperele = (numelements / TET_BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	HANDLE_ERROR( hipMemcpy(gpuptr_TetData, gpuElements, numblocksperele * sizeof(TetData), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_NodeData, gpuNodes, numblockpernode * sizeof(NodeData), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_xt, x, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_x, x, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_x0, x0, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_fext, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_allowed, allowed, numnodes * sizeof(char), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_mass, mass, numnodes * sizeof(float), hipMemcpyHostToDevice) );

	checkCudaErrors("Upload vars");
}

__host__
void
gpuDownloadVars(float* x, int numnodes)
{
	hipMemcpy(x, gpuptr_x, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void inspectGPUBuffer(float* gpubuf,int numnodes)
{
	hipMemcpy(debugbuf, gpubuf, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0;i<numnodes * 3;i++)
		printf("%f\n",debugbuf[i]);
	printf("\n");
	//system("pause");
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_fext, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

	checkCudaErrors("Upload Ext force");
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptr_TetData);
	hipFree(gpuptr_NodeData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_fext);
	hipFree(gpuptr_sn);
	hipFree(gpuptr_x);
	hipFree(gpuptr_x0);
	hipFree(gpuptr_mass);
	hipFree(gpuptr_b);
	hipFree(gpuptr_CG_R);
	hipFree(gpuptr_CG_D);
	hipFree(gpuptr_CG_Q);
	hipFree(gpuptr_CG_Vars);
	hipFree(gpuptr_debug);
}


//ref function
__device__
void readWarpCoalescedMemory(WarpData* data, int node, float out[16])
{
	int warpthreadid = threadIdx.x % HWARP_SIZE;
	int warpdata_tid = (node % (TET_NUM_HWARPS * TET_PER_HWARP));
	int warpdata_bid = (node / (TET_NUM_HWARPS * TET_PER_HWARP));
	int warpdata_wid = warpdata_tid % TET_NUM_HWARPS;
	int warpdata_lid = warpdata_tid / TET_NUM_HWARPS;

	out[warpthreadid] = data[warpdata_bid].P[warpdata_lid][warpdata_wid][warpthreadid];
}

//ref function
__device__
void writeWarpCoalescedMemory(WarpData* data, int node, float out[16])
{
	int warpthreadid = threadIdx.x % HWARP_SIZE;
	int warpdata_tid = (node % (TET_NUM_HWARPS * TET_PER_HWARP));
	int warpdata_bid = (node / (TET_NUM_HWARPS * TET_PER_HWARP));
	int warpdata_wid = warpdata_tid % TET_NUM_HWARPS;
	int warpdata_lid = warpdata_tid / TET_NUM_HWARPS;

	data[warpdata_bid].P[warpdata_lid][warpdata_wid][warpthreadid] = out[warpthreadid];
}


//ref function
// (numnodes / NODE_BLOCK_SIZE) + 1 blocks
__global__
void PDCompressed3x3MatrixMultiply(NodeData* nodedata, float* in, float* out, int max_entry, int numnodes)
{
	
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + threadIdx.x;
	
	float x[3], o[3] = {0};

	if(nodeno < numnodes)
	{
		NodeData* node = &(nodedata[blockIdx.x]);

		x[0] = in[nodeno];
		x[1] = in[nodeno + numnodes];
		x[2] = in[nodeno + numnodes * 2];

		for(int i=0;i<max_entry;i++)
		{
			float entry = node->nodeEntries[i][threadIdx.x];
			o[0] += entry * x[0];
			o[1] += entry * x[1];
			o[2] += entry * x[2];
		}

		out[nodeno] = o[0];
		out[nodeno + numnodes] = o[1];
		out[nodeno + numnodes * 2] = o[2];

	}
}

//1 block, DOT_BLOCK_SIZE
__device__
void dot(const float* a, const float* b, float*  out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>32)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		i>>=1;
	}


	if(index == 0)
		*out = temp[0];
}

__global__
void
MakeSnAndV(float* sn, float* x, float* xt, float* vt, float* fext, float* nodemass, int numnodes)
{
	int nodeno = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(nodeno < numnodes)
	{
	
		float _xt[3];
		float _vt[3];
		float _x[3];

		int mass = nodemass[nodeno];

		_x[0] = x[nodeno];
		_x[1] = x[nodeno + numnodes];
		_x[2] = x[nodeno + numnodes * 2];
		
		_xt[0] = xt[nodeno];
		_xt[1] = xt[nodeno + numnodes];
		_xt[2] = xt[nodeno + numnodes * 2];
		
		_vt[0] = (_x[0] - _xt[0])/DT;
		_vt[1] = (_x[1] - _xt[1])/DT;
		_vt[2] = (_x[2] - _xt[2])/DT; 

		sn[nodeno] =				 (_x[0] + DT * _vt[0] * DAMPING + DT * DT * fext[nodeno] / mass) * (mass/(DT * DT));
		sn[nodeno + numnodes] =		 (_x[1] + DT * _vt[1] * DAMPING + DT * DT * fext[nodeno + numnodes] / mass) * (mass/(DT * DT));
		sn[nodeno + numnodes * 2] =  (_x[2] + DT * _vt[2] * DAMPING + DT * DT * fext[nodeno + numnodes * 2] / mass) * (mass/(DT * DT));

		xt[nodeno] =				_x[0];
		xt[nodeno + numnodes] =		_x[1];
		xt[nodeno + numnodes * 2] = _x[2];

		vt[nodeno] =				_vt[0];
		vt[nodeno + numnodes] =		_vt[1];
		vt[nodeno + numnodes * 2] = _vt[2];

	}
}



// (numtets / TET_BLOCK_SIZE) + 1 blocks
__global__
void ProjectTransforms(TetData* tets, float* x, int numtets, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x *TET_BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numtets)
	{
		TetData* t_ele = &(tets[bid]);
		
		float nodes[12], R[3][3]={0}, D[3][3]={0};

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			int index = t_ele->index[i][ltid];
			nodes[i * 3] = x[index];
			nodes[i * 3 + 1] = x[index + numnodes];
			nodes[i * 3 + 2] = x[index + numnodes * 2];
		}
		
		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				D[i][j] = t_ele->Bm[i][j][ltid];

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
					R[i][j] = (nodes[i] - nodes[9 + i]) * D[0][j] + 
							  (nodes[3 + i] - nodes[9 + i]) * D[1][j] + 
							  (nodes[6 + i] - nodes[9 + i]) * D[2][j];

		gpuComputePolarDecomposition(R);


		float det = R[0][0] * (R[1][1] * R[2][2] - R[1][2] * R[2][1]); 
		det -= R[0][1] * (R[2][2] * R[1][0] - R[1][2] * R[2][0]); 
		det += R[0][2] * (R[1][0] * R[2][1] - R[1][1] * R[2][0]);

		int neg = ((det < 0) * -1) | 1;

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
					R[i][j] = neg * R[i][j];

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				D[i][j] = t_ele->Dm[i][j][ltid];

		float w = t_ele->weight[ltid];

		#pragma unroll 3
		for(int i=0;i<3;i++)
		{
			#pragma unroll 3
			for(int j=0;j<3;j++)
			{
				float temp = 0;

				#pragma unroll 3	
				for(int k=0;k<3;k++)
				{
					temp += R[i][k] * D[k][j];
				}

				nodes[j * 3 + i] = temp * w;
			}
		}

		nodes[9] =  -nodes[0]-nodes[3]-nodes[6];
		nodes[10] = -nodes[1]-nodes[4]-nodes[7];
		nodes[11] = -nodes[2]-nodes[5]-nodes[8];

		t_ele->P[0][0][ltid] = nodes[0];
		t_ele->P[0][1][ltid] = nodes[1];
		t_ele->P[0][2][ltid] = nodes[2];
		t_ele->P[1][0][ltid] = nodes[3];
		t_ele->P[1][1][ltid] = nodes[4];
		t_ele->P[1][2][ltid] = nodes[5];
		t_ele->P[2][0][ltid] = nodes[6];
		t_ele->P[2][1][ltid] = nodes[7];
		t_ele->P[2][2][ltid] = nodes[8];
		t_ele->P[3][0][ltid] = nodes[9];
		t_ele->P[3][1][ltid] = nodes[10];
		t_ele->P[3][2][ltid] = nodes[11];
	}
}

__global__
void makeBandRandD(TetData* tetdata, NodeData* nodedata, float* sn, float* b, int numnodes)
{
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + threadIdx.x;
	int ltid = threadIdx.x;
	int bid = blockIdx.x;

	if(nodeno < numnodes)
	{
		NodeData* nd = &(nodedata[bid]);
		
		float o[3] = {0};

		int n = nd->ntets[ltid];

		o[0] = sn[nodeno];
		o[1] = sn[nodeno + numnodes];
		o[2] = sn[nodeno + numnodes * 2];

		for(int i=0;i<n;i++)
		{
			int tetindex = nd->indices[i][TET_INDEX][ltid];
			int tetblockindex = tetindex / TET_BLOCK_SIZE;
			int tetthreadindex = tetindex % TET_BLOCK_SIZE;
			int nodeindex = nd->indices[i][NODE_INDEX][ltid];
			
			o[0] +=  tetdata[tetblockindex].P[nodeindex][0][tetthreadindex];
			o[1] +=  tetdata[tetblockindex].P[nodeindex][1][tetthreadindex];
			o[2] +=  tetdata[tetblockindex].P[nodeindex][2][tetthreadindex];

		}

		// now: o = b
		b[nodeno] = o[0];
		b[nodeno + numnodes] = o[1];
		b[nodeno + numnodes * 2] = o[2];
		
	}
}

__global__
void makeRandD(NodeData* nodedata, float* b, float* x, float* r, float* d, int max_entry, int numnodes)
{
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + threadIdx.x;
	int ltid = threadIdx.x;
	int bid = blockIdx.x;

	

	if(nodeno < numnodes)
	{
		NodeData* nd = &(nodedata[bid]);
		float o[3];

		o[0] = b[nodeno];
		o[1] = b[nodeno + numnodes];
		o[2] = b[nodeno + numnodes * 2];

		//o = r = d = b - Ax
		for(int i=0;i<max_entry;i++)
		{
			float entry = nd->nodeEntries[i][threadIdx.x];
			int index = nd->nodeEntryIndex[i][threadIdx.x];

			o[0] -= entry * x[index];
			o[1] -= entry * x[index + numnodes];
			o[2] -= entry *  x[index + numnodes * 2];
		}

		r[nodeno] = o[0];
		r[nodeno + numnodes] = o[1];
		r[nodeno + numnodes * 2] = o[2];

		d[nodeno] = o[0];
		d[nodeno + numnodes] = o[1];
		d[nodeno + numnodes * 2] = o[2];
	}
}

//init
//1 block, DOT_BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//LOOP
//make q
__global__
void MakeQ(NodeData* nodedata, float* d, float* q, int max_entry, int numnodes)
{
	
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + threadIdx.x;
	
	float _d[3], o[3] = {0};

	if(nodeno < numnodes)
	{
		NodeData* node = &(nodedata[blockIdx.x]);

		for(int i=0;i<max_entry;i++)
		{
			float entry = node->nodeEntries[i][threadIdx.x];
			int index = node->nodeEntryIndex[i][threadIdx.x];

			_d[0] = d[index];
			_d[1] = d[index + numnodes];
			_d[2] = d[index + numnodes * 2];

			o[0] += entry * _d[0];
			o[1] += entry * _d[1];
			o[2] += entry * _d[2];
		}

		q[nodeno] = o[0];
		q[nodeno + numnodes] = o[1];
		q[nodeno + numnodes * 2] = o[2];

	}
}

__global__
void
makeVars(CGVars* vars, float* d, float* q, float*  r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

__global__
void
makeXRandD(CGVars* vars, float * x, float*  r, float*  d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;
		float d1,d2,d3;
		float r1,r2,r3;

		d1 = d[tid];
		d2 =  d[tid + numnodes];
		d3 = d[tid + numnodes * 2];

		x[tid] = x[tid] + alpha * d1;
		x[tid + numnodes] = x[tid + numnodes] + alpha * d2;
		x[tid + numnodes * 2] = x[tid + numnodes * 2] + alpha * d3;

		r1 = r[tid] - alpha * q[tid];
		r2 = r[tid + numnodes] - alpha * q[tid + numnodes];
		r3 = r[tid + numnodes * 2] - alpha * q[tid + numnodes * 2];

		d[tid] = r1 + beta * d1;
		d[tid + numnodes] = r2 + beta * d2;
		d[tid + numnodes * 2] = r3 + beta * d3;

		r[tid] = r1;
		r[tid + numnodes] = r2;
		r[tid + numnodes * 2] = r3;
	}
} 

__global__
void constrainNodes(float* x, float* x0, char* constrained, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		if(constrained[tid])
		{
			x[tid] = x0[tid];
			x[tid + numnodes] = x0[tid + numnodes];
			x[tid + numnodes * 2] = x0[tid + numnodes * 2];
		}
	}
}


__host__
void GPUTimestep(int numtets, int numnodes, int max_entry)
{
	const int num_blocks_tet = (numtets/TET_BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	//printf("Started\n");
	
	MakeSnAndV<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_sn, gpuptr_x, gpuptr_xt, gpuptr_vt, gpuptr_fext, gpuptr_mass, numnodes);

	//checkCudaErrors("Make Sn, V");

	for(int i=0;i<PD_ITERATIONS;i++)
	{
		ProjectTransforms<<<num_blocks_tet, TET_BLOCK_SIZE>>>(gpuptr_TetData, gpuptr_x, numtets, numnodes);

		//checkCudaErrors("Project Transforms");

		makeBandRandD<<<num_blocks_node, NODE_BLOCK_SIZE>>>(gpuptr_TetData, gpuptr_NodeData, gpuptr_sn, gpuptr_b, numnodes);

		//checkCudaErrors("B, R, D");

		makeRandD<<<num_blocks_node, NODE_BLOCK_SIZE>>>(gpuptr_NodeData, gpuptr_b, gpuptr_x, gpuptr_CG_R, gpuptr_CG_D,max_entry, numnodes);

		//checkCudaErrors("R, D");

		initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptr_CG_Vars, gpuptr_CG_R, numnodes);

		//checkCudaErrors("init CG Vars");

		for(int j=0;j<CG_ITERATIONS;j++)
		{
			MakeQ<<<num_blocks_node, NODE_BLOCK_SIZE>>>(gpuptr_NodeData, gpuptr_CG_D, gpuptr_CG_Q, max_entry, numnodes);

			//checkCudaErrors("q = Ad");

			makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptr_CG_Vars, gpuptr_CG_D, gpuptr_CG_Q, gpuptr_CG_R, numnodes);

			//checkCudaErrors("make CG vars");

			makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_CG_Vars, gpuptr_x, gpuptr_CG_R, gpuptr_CG_D, gpuptr_CG_Q, numnodes);

			//checkCudaErrors("X, R, D");
		}

		constrainNodes<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_x, gpuptr_x0, gpuptr_allowed, numnodes);
	}

}